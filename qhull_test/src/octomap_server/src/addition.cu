#include "hip/hip_runtime.h"
#include <addition.h>
 __global__ void add(int *a, int *b, int *c)
 {
     *c=*a+*b;
 }
 
 extern "C" bool addition(int a, int b, int *c)
 {
     int *d_a, *d_b, *d_c;
     int size=sizeof(int);
     
     hipMalloc((void **)&d_a, size);
     hipMalloc((void **)&d_b, size);
     hipMalloc((void **)&d_c, size);
     
     hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
     hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
     
     add<<<1,1>>>(d_a, d_b, d_c);
     
     hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
     
     hipFree(d_a); hipFree(d_b); hipFree(d_c);
     
     return true;
}